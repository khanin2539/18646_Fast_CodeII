#include "hip/hip_runtime.h"
/*
    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "matrix_mul.h"
#define BLOCK_WIDTH 2

namespace cuda
{
  __global__ void matrix_mul_kernel(float *sq_matrix_1, float *sq_matrix_2, float *sq_matrix_result, int sq_dimension)
  {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    for(int k = 0; k < sq_dimension; k++)
    {	
	sq_matrix_result[row*sq_dimension + col] += sq_matrix_1[row * sq_dimension + k] * sq_matrix_2[k * sq_dimension + col];
    }
  }
  
  void matrix_multiplication(float *sq_matrix_1, float *sq_matrix_2, float *sq_matrix_result, unsigned int sq_dimension)
  {
    int size = sq_dimension * sq_dimension * sizeof(float);
    float *sq_matrix_1_d, *sq_matrix_2_d, *sq_matrix_result_d;
    
    /***************************************************
    Step 1: Allocation of memory on device memory  
    ****************************************************/
    
    /* copy sq_matrix_1 and sq_matrix_2 to device memory */
    hipMalloc((void**) &sq_matrix_1_d, size);
    hipMemcpy(sq_matrix_1_d, sq_matrix_1, size, hipMemcpyHostToDevice);
    hipMalloc((void**) &sq_matrix_2_d, size);
    hipMemcpy(sq_matrix_2_d, sq_matrix_2, size, hipMemcpyHostToDevice);
    
    /*allocate sq_matrix_result on host */
    hipMalloc((void**) &sq_matrix_result_d, size);
    
    /***************************************************
    Step 2: Invoke kernel 
    ****************************************************/
    int blockNum = ceil(sq_dimension * 1.0 / BLOCK_WIDTH);
    dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);
    dim3 dimGrid(blockNum, blockNum);

    matrix_mul_kernel<<<dimGrid, dimBlock>>>(sq_matrix_1_d, sq_matrix_2_d, sq_matrix_result_d, sq_dimension);
    
    /***************************************************
    Step 3: Transfer result from device to host 
    ****************************************************/
    hipMemcpy(sq_matrix_result, sq_matrix_result_d, size, hipMemcpyDeviceToHost);
    hipFree(sq_matrix_1_d);
    hipFree(sq_matrix_2_d);
    hipFree(sq_matrix_result_d);
  }  
} // namespace cuda
