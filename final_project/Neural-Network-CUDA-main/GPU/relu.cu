#include "hip/hip_runtime.h"
#include "relu.h"


__global__
void relu_forward_gpu(float *inp, float *out, int sz_out){
    int ind = blockDim.x*blockIdx.x + threadIdx.x;
    
    if (ind < sz_out){
        out[ind] = fmaxf(0, inp[ind]);
    }
}


__global__
void relu_backward_gpu(float *inp, float *out, int sz_out){
    int ind = blockDim.x*blockIdx.x + threadIdx.x;
    
    if (ind < sz_out){
        inp[ind] = (0 < inp[ind]) * out[ind];
    }
}


ReLU_GPU::ReLU_GPU(int _sz_out){
    sz_out = _sz_out;
    
    n_blocks = (sz_out + 1024 - 1) / 1024;
}


void ReLU_GPU::forward(float *_inp, float *_out){
    inp = _inp;
    out = _out;

    relu_forward_gpu<<<n_blocks, 1024>>>(inp, out, sz_out);
    hipDeviceSynchronize();
}


void ReLU_GPU::backward(){    
    relu_backward_gpu<<<n_blocks, 1024>>>(inp, out, sz_out);
    hipDeviceSynchronize();
}
